﻿// covid_simulation_complete.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "gpu_define.cuh"
#include "gpu_person.cuh"
#include "gpu_utils.cuh"
#include "gpu_aleat.cuh"
#include "gpu_begin.cuh"
#include "gpu_neighbors.cuh"
#include "update_kernel.cuh"
#include "gpu_update_boundaries.cuh"

// Include all state kernels
#include "S_kernel.cuh"
#include "E_kernel.cuh"
#include "IP_kernel.cuh"
#include "IS_kernel.cuh"
#include "H_kernel.cuh"
#include "ICU_kernel.cuh"

// Function to run one simulation day
void runSimulationDay(GPUPerson* d_population, unsigned int* d_rngStates,
    int L, int day, int blockSize, int numBlocks) {

    printf("\n--- Day %d ---\n", day);

    // Update boundaries
    printf("Updating boundaries...\n");
    updateBoundaries_kernel << <numBlocks, blockSize >> > (d_population, L);
    hipDeviceSynchronize();

    // Run state kernels
    printf("Running S kernel...\n");
    S_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    printf("Running E kernel...\n");
    E_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    printf("Running IP kernel...\n");
    IP_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    printf("Running IS kernel...\n");
    IS_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    printf("Running H kernel...\n");
    H_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    printf("Running ICU kernel...\n");
    ICU_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    // Reset counters for update kernel
    resetCounters_kernel << <1, 1 >> > ();
    resetNewCounters_kernel << <1, 1 >> > ();
    hipDeviceSynchronize();

    // Run update kernel
    printf("Running update kernel...\n");
    update_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L, day, d_ProbNaturalDeath);
    hipDeviceSynchronize();

    // Get statistics
    int h_totals[15] = { 0 };
    int h_new_cases[15] = { 0 };
    getCountersFromDevice(h_totals, h_new_cases);

    // Print key statistics
    printf("Day %d statistics:\n", day);
    printf("  S: %d (new: %d)\n", h_totals[S], h_new_cases[S]);
    printf("  E: %d (new: %d)\n", h_totals[E], h_new_cases[E]);
    printf("  IP: %d (new: %d)\n", h_totals[IP], h_new_cases[IP]);
    printf("  IA: %d (new: %d)\n", h_totals[IA], h_new_cases[IA]);
    printf("  IS (L/M/S): %d/%d/%d\n", h_totals[ISLight], h_totals[ISModerate], h_totals[ISSevere]);
    printf("  H: %d, ICU: %d\n", h_totals[H], h_totals[ICU]);
    printf("  Recovered: %d, Deaths: %d\n", h_totals[Recovered], h_totals[DeadCovid]);

    // Reset new counters for next day
    resetNewCounters_kernel << <1, 1 >> > ();
    hipDeviceSynchronize();
}

int main(int argc, char* argv[]) {
    printf("COVID-19 CUDA Simulation - Complete Version\n");

    // Initialize city and GPU constants
    int city = ROC;  // Rocinha
    setupCityParameters(city);
    setupGPUConstants();

    // Simulation parameters
    int simulationNumber = 1;
    const int L = 632;  // Grid size
    const int gridSize = (L + 2) * (L + 2);
    const int N = L * L;
    const int DAYS_TO_RUN = 200;  // Run for 10 days as a test

    printf("Grid size: %d x %d = %d cells\n", L, L, N);
    printf("Running for %d days\n", DAYS_TO_RUN);

    // Allocate device memory
    GPUPerson* d_population;
    hipMalloc(&d_population, gridSize * sizeof(GPUPerson));

    // Initialize RNG
    unsigned int* d_rngStates;
    hipMalloc(&d_rngStates, gridSize * sizeof(unsigned int));

    unsigned int seed = 893221891 * simulationNumber;
    int blockSize = 256;
    int numBlocks = (gridSize + blockSize - 1) / blockSize;

    initRNG << <numBlocks, blockSize >> > (d_rngStates, seed, gridSize);
    hipDeviceSynchronize();

    // Initialize population
    printf("\nInitializing population...\n");
    initPopulation_kernel << <numBlocks, blockSize >> > (d_population, d_rngStates, L);
    hipDeviceSynchronize();

    // Initialize counters
    int* d_stateCounts, * d_newCounts;
    hipMalloc(&d_stateCounts, 15 * sizeof(int));
    hipMalloc(&d_newCounts, 15 * sizeof(int));

    initCounters_kernel << <1, 32 >> > (d_stateCounts, d_newCounts, N);
    hipDeviceSynchronize();

    // Distribute initial infections
    printf("Distributing initial infections (5 IP cases)...\n");
    distributeInitialInfections_kernel << <1, 1 >> > (
        d_population, d_rngStates, d_stateCounts, d_newCounts, L,
        0,  // Eini
        5,  // IPini
        0,  // IAini
        0,  // ISLightini
        0,  // ISModerateini
        0   // ISSevereini
        );
    hipDeviceSynchronize();

    // Set available beds
    int availableBeds = NumberOfHospitalBeds - NumberOfHospitalBeds * AverageOcupationRateBeds;
    int availableBedsICU = NumberOfICUBeds - NumberOfICUBeds * AverageOcupationRateBedsICU;
    hipMemcpyToSymbol(HIP_SYMBOL(AvailableBeds), &availableBeds, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(AvailableBedsICU), &availableBedsICU, sizeof(int));

    printf("\nStarting simulation...\n");
    printf("Available beds: Hospital=%d, ICU=%d\n", availableBeds, availableBedsICU);

    // Run simulation for specified days
    for (int day = 1; day <= DAYS_TO_RUN; day++) {
        runSimulationDay(d_population, d_rngStates, L, day, blockSize, numBlocks);
    }

    // Final statistics
    printf("\n=== Final Statistics ===\n");
    int h_totals[15] = { 0 };
    int h_new_cases[15] = { 0 };
    getCountersFromDevice(h_totals, h_new_cases);

    int totalInfectious = h_totals[ISLight] + h_totals[ISModerate] + h_totals[ISSevere];
    int totalPopulation = 0;
    for (int i = 0; i < 15; i++) {
        totalPopulation += h_totals[i];
    }

    printf("Total population check: %d (should be close to %d)\n", totalPopulation, N);
    printf("Total infectious: %d\n", totalInfectious);
    printf("Total recovered: %d\n", h_totals[Recovered]);
    printf("Total COVID deaths: %d\n", h_totals[DeadCovid]);

    // Cleanup
    printf("\nCleaning up...\n");
    hipFree(d_population);
    hipFree(d_rngStates);
    hipFree(d_stateCounts);
    hipFree(d_newCounts);
    cleanupGPUConstants();

    printf("\nSimulation completed successfully!\n");

    return 0;
}